#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#include "reduction.h"

void allocateDeviceMemory(void** M, int size)
{
    hipError_t err = hipMalloc(M, size);
    assert(err==hipSuccess);
}


void deallocateDeviceMemory(void* M)
{
    hipError_t err = hipFree(M);
    assert(err==hipSuccess);
}

void cudaMemcpyToDevice(void* dst, void* src, int size) {
    hipError_t err = hipMemcpy((void*)dst, (void*)src, size, hipMemcpyHostToDevice);
    assert(err==hipSuccess);
}

void cudaMemcpyToHost(void* dst, void* src, int size) {
    hipError_t err = hipMemcpy((void*)dst, (void*)src, size, hipMemcpyDeviceToHost);
    assert(err==hipSuccess);
}

void reduce_ref(const int* const g_idata, int* const g_odata, const int n) {
    for (int i = 0; i < n; i++)
        g_odata[0] += g_idata[i];
}


void reduce_optimize(const int* const g_idata, int* const g_odata, const int* const d_idata, int* const d_odata, const int n) {
    // TODO: Implement your CUDA code
    // Reduction result must be stored in d_odata[0] 
    // You should run the best kernel in here but you must remain other kernels as evidence.
}
